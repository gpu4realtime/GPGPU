#include "hip/hip_runtime.h"
#include "public.h"

#include "helper.h"
#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <cassert>

#define TILE_DIM   32
#define MAX_N_REG  32
#define NUM_OF_WARPS_IN_BLOCK 8

namespace mat_gpu
{
	template<typename dataT>
	struct Data
	{
		dataT x_00_0 = 0;
		dataT x_00_1 = 0;
		dataT x_00_2 = 0;
		dataT x_00_3 = 0;

		dataT x_01_0 = 0;
		dataT x_01_1 = 0;
		dataT x_01_2 = 0;
		dataT x_01_3 = 0;

		dataT x_10_0 = 0;
		dataT x_10_1 = 0;
		dataT x_10_2 = 0;
		dataT x_10_3 = 0;

		dataT x_11_0 = 0;
		dataT x_11_1 = 0;
		dataT x_11_2 = 0;
		dataT x_11_3 = 0;
	};

	template<typename dataT>
	__device__ void compute_echelon_and_row_reduced_echelon_form_generic( Data<dataT>& data
									    , dataT shared_00[TILE_DIM]
									    , dataT shared_01[TILE_DIM]
									    , int i
									    , int j)
	{
		#pragma unroll TILE_DIM
		for (int k = 0; k < TILE_DIM; ++k)
		{
			dataT x_00_p = shared_00[j];
			dataT x_01_p = shared_01[j];

			dataT num = __shfl_sync(0xFFFFFFFF, data.x_00_0, k, TILE_DIM);

			data.x_00_0 = (4 * i <= k || j <= k) ? data.x_00_0 : __fadd_rz(data.x_00_0, -__fmul_rz(num, x_00_p));
			data.x_01_0 = 4 * i <= k ? data.x_01_0 : __fadd_rz(data.x_01_0, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_00_1, k, TILE_DIM);

			data.x_00_1 = (4 * i + 1 <= k || j <= k) ? data.x_00_1 : __fadd_rz(data.x_00_1, -__fmul_rz(num, x_00_p));
			data.x_01_1 = 4 * i + 1 <= k ? data.x_01_1 : __fadd_rz(data.x_01_1, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_00_2, k, TILE_DIM);

			data.x_00_2 = (4 * i + 2 <= k || j <= k) ? data.x_00_2 : __fadd_rz(data.x_00_2, -__fmul_rz(num, x_00_p));
			data.x_01_2 = 4 * i + 2 <= k ? data.x_01_2 : __fadd_rz(data.x_01_2, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_00_3, k, TILE_DIM);

			data.x_00_3 = (4 * i + 3 <= k || j <= k) ? data.x_00_3 : __fadd_rz(data.x_00_3, -__fmul_rz(num, x_00_p));
			data.x_01_3 = 4 * i + 3 <= k ? data.x_01_3 : __fadd_rz(data.x_01_3, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_10_0, k, TILE_DIM);

			data.x_10_0 = j <= k ? data.x_10_0 : __fadd_rz(data.x_10_0, -__fmul_rz(num, x_00_p));
			data.x_11_0 = __fadd_rz(data.x_11_0, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_10_1, k, TILE_DIM);

			data.x_10_1 = j <= k ? data.x_10_1 : __fadd_rz(data.x_10_1, -__fmul_rz(num, x_00_p));
			data.x_11_1 = __fadd_rz(data.x_11_1, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_10_2, k, TILE_DIM);

			data.x_10_2 = j <= k ? data.x_10_2 : __fadd_rz(data.x_10_2, -__fmul_rz(num, x_00_p));
			data.x_11_2 = __fadd_rz(data.x_11_2, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_10_3, k, TILE_DIM);

			data.x_10_3 = j <= k ? data.x_10_3 : __fadd_rz(data.x_10_3, -__fmul_rz(num, x_00_p));
			data.x_11_3 = __fadd_rz(data.x_11_3, -__fmul_rz(num, x_01_p));

			__syncthreads();

			if (k + 1 == 4 * i)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_0, k + 1, TILE_DIM);

				data.x_00_0 = __fdiv_rz(data.x_00_0, val);
				data.x_01_0 = __fdiv_rz(data.x_01_0, val);

				shared_00[j] = data.x_00_0;
				shared_01[j] = data.x_01_0;
			}
			else if (k + 1 == 4 * i + 1)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_1, k + 1, TILE_DIM);

				data.x_00_1 = __fdiv_rz(data.x_00_1, val);
				data.x_01_1 = __fdiv_rz(data.x_01_1, val);

				shared_00[j] = data.x_00_1;
				shared_01[j] = data.x_01_1;
			}
			else if (k + 1 == 4 * i + 2)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_2, k + 1, TILE_DIM);

				data.x_00_2 = __fdiv_rz(data.x_00_2, val);
				data.x_01_2 = __fdiv_rz(data.x_01_2, val);

				shared_00[j] = data.x_00_2;
				shared_01[j] = data.x_01_2;
			}
			else if (k + 1 == 4 * i + 3)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_3, k + 1, TILE_DIM);

				data.x_00_3 = __fdiv_rz(data.x_00_3, val);
				data.x_01_3 = __fdiv_rz(data.x_01_3, val);

				shared_00[j] = data.x_00_3;
				shared_01[j] = data.x_01_3;
			}

			__syncthreads();
		}
	}

	template<typename dataT>
	__device__ void compute_echelon_and_row_reduced_echelon_form_row( Data<dataT>& data
									, dataT shared_00[TILE_DIM]
									, dataT shared_01[TILE_DIM]
									, int i
									, int j)
	{
		#pragma unroll TILE_DIM
		for (int k = 0; k < TILE_DIM; ++k)
		{
			dataT x_00_p = shared_00[j];
			dataT x_01_p = shared_01[j];

			dataT num = __shfl_sync(0xFFFFFFFF, data.x_00_0, k, TILE_DIM);

			data.x_00_0 = (4 * i == k || j <= k) ? data.x_00_0 : __fadd_rz(data.x_00_0, -__fmul_rz(num, x_00_p));
			data.x_01_0 = (4 * i == k) ? data.x_01_0 : __fadd_rz(data.x_01_0, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_00_1, k, TILE_DIM);

			data.x_00_1 = (4 * i + 1 == k || j <= k) ? data.x_00_1 : __fadd_rz(data.x_00_1, -__fmul_rz(num, x_00_p));
			data.x_01_1 = (4 * i + 1 == k) ? data.x_01_1 : __fadd_rz(data.x_01_1, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_00_2, k, TILE_DIM);

			data.x_00_2 = (4 * i + 2 == k || j <= k) ? data.x_00_2 : __fadd_rz(data.x_00_2, -__fmul_rz(num, x_00_p));
			data.x_01_2 = (4 * i + 2 == k) ? data.x_01_2 : __fadd_rz(data.x_01_2, -__fmul_rz(num, x_01_p));

			num = __shfl_sync(0xFFFFFFFF, data.x_00_3, k, TILE_DIM);

			data.x_00_3 = (4 * i + 3 == k || j <= k) ? data.x_00_3 : __fadd_rz(data.x_00_3, -__fmul_rz(num, x_00_p));
			data.x_01_3 = (4 * i + 3 == k) ? data.x_01_3 : __fadd_rz(data.x_01_3, -__fmul_rz(num, x_01_p));

			__syncthreads();

			if (k + 1 == 4 * i)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_0, k + 1, TILE_DIM);

				data.x_00_0 = __fdiv_rz(data.x_00_0, val);
				data.x_01_0 = __fdiv_rz(data.x_01_0, val);

				shared_00[j] = data.x_00_0;
				shared_01[j] = data.x_01_0;
			}
			else if (k + 1 == 4 * i + 1)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_1, k + 1, TILE_DIM);

				data.x_00_1 = __fdiv_rz(data.x_00_1, val);
				data.x_01_1 = __fdiv_rz(data.x_01_1, val);

				shared_00[j] = data.x_00_1;
				shared_01[j] = data.x_01_1;
			}
			else if (k + 1 == 4 * i + 2)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_2, k + 1, TILE_DIM);

				data.x_00_2 = __fdiv_rz(data.x_00_2, val);
				data.x_01_2 = __fdiv_rz(data.x_01_2, val);

				shared_00[j] = data.x_00_2;
				shared_01[j] = data.x_01_2;
			}
			else if (k + 1 == 4 * i + 3)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_3, k + 1, TILE_DIM);

				data.x_00_3 = __fdiv_rz(data.x_00_3, val);
				data.x_01_3 = __fdiv_rz(data.x_01_3, val);

				shared_00[j] = data.x_00_3;
				shared_01[j] = data.x_01_3;
			}

			__syncthreads();
		}
	}

	template<typename dataT>
	__device__ void compute_echelon_and_row_reduced_echelon_form_pivot( Data<dataT>& data
									  , dataT shared_00[TILE_DIM]
									  , dataT shared_01[TILE_DIM]
									  , int i
									  , int j)
	{
		#pragma unroll TILE_DIM
		for (int k = 0; k < TILE_DIM; ++k)
		{
			data.x_00_0 = (4 * i == k) ? data.x_00_0 : __fadd_rz(data.x_00_0, -__fmul_rz(shared_00[j], __shfl_sync(0xFFFFFFFF, data.x_00_0, k, TILE_DIM)));

			data.x_00_1 = (4 * i + 1 == k) ? data.x_00_1 : __fadd_rz(data.x_00_1, -__fmul_rz(shared_00[j], __shfl_sync(0xFFFFFFFF, data.x_00_1, k, TILE_DIM)));

			data.x_00_2 = (4 * i + 2 == k) ? data.x_00_2 : __fadd_rz(data.x_00_2, -__fmul_rz(shared_00[j], __shfl_sync(0xFFFFFFFF, data.x_00_2, k, TILE_DIM)));

			data.x_00_3 = (4 * i + 3 == k) ? data.x_00_3 : __fadd_rz(data.x_00_3, -__fmul_rz(shared_00[j], __shfl_sync(0xFFFFFFFF, data.x_00_3, k, TILE_DIM)));

			__syncthreads();

			if (k + 1 == 4 * i)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_0, k + 1, TILE_DIM);

				data.x_00_0 = __fdiv_rz(data.x_00_0, val);

				shared_00[j] = data.x_00_0;
			}
			else if (k + 1 == 4 * i + 1)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_1, k + 1, TILE_DIM);

				data.x_00_1 = __fdiv_rz(data.x_00_1, val);

				shared_00[j] = data.x_00_1;
			}
			else if (k + 1 == 4 * i + 2)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_2, k + 1, TILE_DIM);

				data.x_00_2 = __fdiv_rz(data.x_00_2, val);

				shared_00[j] = data.x_00_2;
			}
			else if (k + 1 == 4 * i + 3)
			{
				dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_3, k + 1, TILE_DIM);

				data.x_00_3 = __fdiv_rz(data.x_00_3, val);

				shared_00[j] = data.x_00_3;
			}

			__syncthreads();
		}
	}

	template<typename dataT>
	__global__ void __maxnreg__(MAX_N_REG) init_and_compute_echelon( dataT* in
								       , dataT* augmat
								       , dataT* buffer
								       , int n
								       , int nblocks
								       , int iter)
	{
		__shared__ dataT shared_00[TILE_DIM];
		__shared__ dataT shared_01[TILE_DIM];

		int block_col = blockIdx.x / nblocks + 1;
		int block_row = blockIdx.x - nblocks * (blockIdx.x / nblocks);

		Data<dataT> data;

		data.x_00_0 = in[(TILE_DIM * iter + 4 * threadIdx.y) * n + (TILE_DIM * iter + threadIdx.x)];
		data.x_00_1 = in[(TILE_DIM * iter + 4 * threadIdx.y + 1) * n + (TILE_DIM * iter + threadIdx.x)];
		data.x_00_2 = in[(TILE_DIM * iter + 4 * threadIdx.y + 2) * n + (TILE_DIM * iter + threadIdx.x)];
		data.x_00_3 = in[(TILE_DIM * iter + 4 * threadIdx.y + 3) * n + (TILE_DIM * iter + threadIdx.x)];

		if (block_col < nblocks)
		{
			data.x_01_0 = in[(TILE_DIM * iter + 4 * threadIdx.y) * n + (TILE_DIM * block_col + threadIdx.x)];
			data.x_01_1 = in[(TILE_DIM * iter + 4 * threadIdx.y + 1) * n + (TILE_DIM * block_col + threadIdx.x)];
			data.x_01_2 = in[(TILE_DIM * iter + 4 * threadIdx.y + 2) * n + (TILE_DIM * block_col + threadIdx.x)];
			data.x_01_3 = in[(TILE_DIM * iter + 4 * threadIdx.y + 3) * n + (TILE_DIM * block_col + threadIdx.x)];

			data.x_11_0 = in[(TILE_DIM * block_row + 4 * threadIdx.y) * n + (TILE_DIM * block_col + threadIdx.x)];
			data.x_11_1 = in[(TILE_DIM * block_row + 4 * threadIdx.y + 1) * n + (TILE_DIM * block_col + threadIdx.x)];
			data.x_11_2 = in[(TILE_DIM * block_row + 4 * threadIdx.y + 2) * n + (TILE_DIM * block_col + threadIdx.x)];
			data.x_11_3 = in[(TILE_DIM * block_row + 4 * threadIdx.y + 3) * n + (TILE_DIM * block_col + threadIdx.x)];
		}
		else
		{
			data.x_01_0 = 4 * threadIdx.y != threadIdx.x ? 0 : 1;
			data.x_01_1 = 4 * threadIdx.y + 1 != threadIdx.x ? 0 : 1;
			data.x_01_2 = 4 * threadIdx.y + 2 != threadIdx.x ? 0 : 1;
			data.x_01_3 = 4 * threadIdx.y + 3 != threadIdx.x ? 0 : 1;
		}

		data.x_10_0 = in[(TILE_DIM * block_row + 4 * threadIdx.y) * n + (TILE_DIM * iter + threadIdx.x)];
		data.x_10_1 = in[(TILE_DIM * block_row + 4 * threadIdx.y + 1) * n + (TILE_DIM * iter + threadIdx.x)];
		data.x_10_2 = in[(TILE_DIM * block_row + 4 * threadIdx.y + 2) * n + (TILE_DIM * iter + threadIdx.x)];
		data.x_10_3 = in[(TILE_DIM * block_row + 4 * threadIdx.y + 3) * n + (TILE_DIM * iter + threadIdx.x)];

		if (threadIdx.y == 0)
		{
			dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_0, 0, TILE_DIM);

			data.x_00_0 = __fdiv_rz(data.x_00_0, val);
			data.x_01_0 = __fdiv_rz(data.x_01_0, val);

			shared_00[threadIdx.x] = data.x_00_0;
			shared_01[threadIdx.x] = data.x_01_0;
		}

		__syncthreads();

		if (blockIdx.x == nblocks * nblocks)
		{
			compute_echelon_and_row_reduced_echelon_form_pivot(data, shared_00, shared_01, threadIdx.y, threadIdx.x);

			augmat[(4 * threadIdx.y + TILE_DIM * iter) * 2 * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_0;
			augmat[(4 * threadIdx.y + 1 + TILE_DIM * iter) * 2 * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_1;
			augmat[(4 * threadIdx.y + 2 + TILE_DIM * iter) * 2 * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_2;
			augmat[(4 * threadIdx.y + 3 + TILE_DIM * iter) * 2 * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_3;

			return;
		}

		if (block_row == iter)
		{
			compute_echelon_and_row_reduced_echelon_form_row(data, shared_00, shared_01, threadIdx.y, threadIdx.x);

			if (block_col == 1)
			{
				block_col = 3;

				buffer[(4 * threadIdx.y + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_01_0;
				buffer[(4 * threadIdx.y + 1 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_01_1;
				buffer[(4 * threadIdx.y + 2 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_01_2;
				buffer[(4 * threadIdx.y + 3 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_01_3;

				return;
			}

			augmat[(4 * threadIdx.y + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_0;
			augmat[(4 * threadIdx.y + 1 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_1;
			augmat[(4 * threadIdx.y + 2 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_2;
			augmat[(4 * threadIdx.y + 3 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_3;

			return;
		}

		compute_echelon_and_row_reduced_echelon_form_generic(data, shared_00, shared_01, threadIdx.y, threadIdx.x);

		if (block_row == iter + 1)
		{
			block_row = 1;

			buffer[(4 * threadIdx.y + TILE_DIM * block_row) * n + (TILE_DIM * (block_col - 1) + threadIdx.x)] = data.x_11_0;
			buffer[(4 * threadIdx.y + 1 + TILE_DIM * block_row) * n + (TILE_DIM * (block_col - 1) + threadIdx.x)] = data.x_11_1;
			buffer[(4 * threadIdx.y + 2 + TILE_DIM * block_row) * n + (TILE_DIM * (block_col - 1) + threadIdx.x)] = data.x_11_2;
			buffer[(4 * threadIdx.y + 3 + TILE_DIM * block_row) * n + (TILE_DIM * (block_col - 1) + threadIdx.x)] = data.x_11_3;

			return;
		}

		if (block_col == iter + 1)
		{
			block_col = 3;

			buffer[(4 * threadIdx.y + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_11_0;
			buffer[(4 * threadIdx.y + 1 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_11_1;
			buffer[(4 * threadIdx.y + 2 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_11_2;
			buffer[(4 * threadIdx.y + 3 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_11_3;

			return;
		}

		augmat[(4 * threadIdx.y + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_0;
		augmat[(4 * threadIdx.y + 1 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_1;
		augmat[(4 * threadIdx.y + 2 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_2;
		augmat[(4 * threadIdx.y + 3 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_3;
	}

	template<typename dataT>
	__global__ void __maxnreg__(MAX_N_REG) compute_echelon( dataT* inv
							      , dataT* augmat
							      , dataT* buffer
							      , int n
							      , int nblocks
							      , int iter)
	{
		__shared__ dataT shared_00[TILE_DIM];
		__shared__ dataT shared_01[TILE_DIM];

		int block_col = blockIdx.x / nblocks + iter + 1;
		int block_row = blockIdx.x - nblocks * (blockIdx.x / nblocks);

		Data<dataT> data;

		data.x_00_0 = buffer[(TILE_DIM * (iter & 1) + 4 * threadIdx.y) * n + threadIdx.x];
		data.x_00_1 = buffer[(TILE_DIM * (iter & 1) + 4 * threadIdx.y + 1) * n + threadIdx.x];
		data.x_00_2 = buffer[(TILE_DIM * (iter & 1) + 4 * threadIdx.y + 2) * n + threadIdx.x];
		data.x_00_3 = buffer[(TILE_DIM * (iter & 1) + 4 * threadIdx.y + 3) * n + threadIdx.x];

		if (block_col < nblocks + iter)
		{
			data.x_01_0 = buffer[(TILE_DIM * (iter & 1) + 4 * threadIdx.y) * n + (TILE_DIM * (block_col - iter) + threadIdx.x)];
			data.x_01_1 = buffer[(TILE_DIM * (iter & 1) + 4 * threadIdx.y + 1) * n + (TILE_DIM * (block_col - iter) + threadIdx.x)];
			data.x_01_2 = buffer[(TILE_DIM * (iter & 1) + 4 * threadIdx.y + 2) * n + (TILE_DIM * (block_col - iter) + threadIdx.x)];
			data.x_01_3 = buffer[(TILE_DIM * (iter & 1) + 4 * threadIdx.y + 3) * n + (TILE_DIM * (block_col - iter) + threadIdx.x)];

			data.x_11_0 = augmat[(TILE_DIM * block_row + 4 * threadIdx.y) * 2 * n + (TILE_DIM * block_col + threadIdx.x)];
			data.x_11_1 = augmat[(TILE_DIM * block_row + 4 * threadIdx.y + 1) * 2 * n + (TILE_DIM * block_col + threadIdx.x)];
			data.x_11_2 = augmat[(TILE_DIM * block_row + 4 * threadIdx.y + 2) * 2 * n + (TILE_DIM * block_col + threadIdx.x)];
			data.x_11_3 = augmat[(TILE_DIM * block_row + 4 * threadIdx.y + 3) * 2 * n + (TILE_DIM * block_col + threadIdx.x)];
		}
		else
		{
			data.x_01_0 = 4 * threadIdx.y != threadIdx.x ? 0 : 1;
			data.x_01_1 = 4 * threadIdx.y + 1 != threadIdx.x ? 0 : 1;
			data.x_01_2 = 4 * threadIdx.y + 2 != threadIdx.x ? 0 : 1;
			data.x_01_3 = 4 * threadIdx.y + 3 != threadIdx.x ? 0 : 1;
		}

		if (block_col != iter || block_row != iter)
		{
			data.x_10_0 = buffer[(TILE_DIM * ((iter & 1) + 2) + 4 * threadIdx.y) * n + (TILE_DIM * block_row + threadIdx.x)];
			data.x_10_1 = buffer[(TILE_DIM * ((iter & 1) + 2) + 4 * threadIdx.y + 1) * n + (TILE_DIM * block_row + threadIdx.x)];
			data.x_10_2 = buffer[(TILE_DIM * ((iter & 1) + 2) + 4 * threadIdx.y + 2) * n + (TILE_DIM * block_row + threadIdx.x)];
			data.x_10_3 = buffer[(TILE_DIM * ((iter & 1) + 2) + 4 * threadIdx.y + 3) * n + (TILE_DIM * block_row + threadIdx.x)];
		}
		else
		{
			data.x_10_0 = data.x_00_0;
			data.x_10_1 = data.x_00_1;
			data.x_10_2 = data.x_00_2;
			data.x_10_3 = data.x_00_3;
		}

		if (threadIdx.y == 0)
		{
			dataT val = __shfl_sync(0xFFFFFFFF, data.x_00_0, 0, TILE_DIM);

			data.x_00_0 = __fdiv_rz(data.x_00_0, val);
			data.x_01_0 = __fdiv_rz(data.x_01_0, val);

			shared_00[threadIdx.x] = data.x_00_0;
			shared_01[threadIdx.x] = data.x_01_0;
		}

		__syncthreads();

		if (blockIdx.x == nblocks * nblocks)
		{
			compute_echelon_and_row_reduced_echelon_form_pivot(data, shared_00, shared_01, threadIdx.y, threadIdx.x);

			if (iter < nblocks - 1)
			{
				augmat[(4 * threadIdx.y + TILE_DIM * iter) * 2 * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_0;
				augmat[(4 * threadIdx.y + 1 + TILE_DIM * iter) * 2 * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_1;
				augmat[(4 * threadIdx.y + 2 + TILE_DIM * iter) * 2 * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_2;
				augmat[(4 * threadIdx.y + 3 + TILE_DIM * iter) * 2 * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_3;

				return;
			}
			
			block_col -= (iter + 1);

			inv[(4 * threadIdx.y + TILE_DIM * iter) * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_0;
			inv[(4 * threadIdx.y + 1 + TILE_DIM * iter) * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_1;
			inv[(4 * threadIdx.y + 2 + TILE_DIM * iter) * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_2;
			inv[(4 * threadIdx.y + 3 + TILE_DIM * iter) * n + (TILE_DIM * iter + threadIdx.x)] = data.x_00_3;

			return;
		}

		if (block_row == iter)
		{
			compute_echelon_and_row_reduced_echelon_form_row(data, shared_00, shared_01, threadIdx.y, threadIdx.x);

			if (iter < (nblocks - 1) && block_col == iter + 1)
			{
				block_col = ((iter - 1) & 1) + 2;

				buffer[(4 * threadIdx.y + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_01_0;
				buffer[(4 * threadIdx.y + 1 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_01_1;
				buffer[(4 * threadIdx.y + 2 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_01_2;
				buffer[(4 * threadIdx.y + 3 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_01_3;

				return;
			}

			if (iter < nblocks - 1)
			{
				augmat[(4 * threadIdx.y + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_0;
				augmat[(4 * threadIdx.y + 1 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_1;
				augmat[(4 * threadIdx.y + 2 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_2;
				augmat[(4 * threadIdx.y + 3 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_3;

				return;
			}

			block_col -= (iter + 1);

			inv[(4 * threadIdx.y + TILE_DIM * block_row) * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_0;
			inv[(4 * threadIdx.y + 1 + TILE_DIM * block_row) * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_1;
			inv[(4 * threadIdx.y + 2 + TILE_DIM * block_row) * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_2;
			inv[(4 * threadIdx.y + 3 + TILE_DIM * block_row) * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_01_3;

			return;
		}

		compute_echelon_and_row_reduced_echelon_form_generic(data, shared_00, shared_01, threadIdx.y, threadIdx.x);

		if (iter < (nblocks - 1) && block_row == iter + 1)
		{
			block_row = ((iter - 1) & 1);

			buffer[(4 * threadIdx.y + TILE_DIM * block_row) * n + (TILE_DIM * (block_col - 1 - iter) + threadIdx.x)] = data.x_11_0;
			buffer[(4 * threadIdx.y + 1 + TILE_DIM * block_row) * n + (TILE_DIM * (block_col - 1 - iter) + threadIdx.x)] = data.x_11_1;
			buffer[(4 * threadIdx.y + 2 + TILE_DIM * block_row) * n + (TILE_DIM * (block_col - 1 - iter) + threadIdx.x)] = data.x_11_2;
			buffer[(4 * threadIdx.y + 3 + TILE_DIM * block_row) * n + (TILE_DIM * (block_col - 1 - iter) + threadIdx.x)] = data.x_11_3;

			return;
		}

		if (iter < (nblocks - 1) && block_col == iter + 1)
		{
			block_col = ((iter - 1) & 1) + 2;

			buffer[(4 * threadIdx.y + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_11_0;
			buffer[(4 * threadIdx.y + 1 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_11_1;
			buffer[(4 * threadIdx.y + 2 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_11_2;
			buffer[(4 * threadIdx.y + 3 + TILE_DIM * block_col) * n + (TILE_DIM * block_row + threadIdx.x)] = data.x_11_3;

			return;
		}

		if (iter < nblocks - 1)
		{
			augmat[(4 * threadIdx.y + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_0;
			augmat[(4 * threadIdx.y + 1 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_1;
			augmat[(4 * threadIdx.y + 2 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_2;
			augmat[(4 * threadIdx.y + 3 + TILE_DIM * block_row) * 2 * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_3;

			return;
		}

		block_col -= (iter + 1);

		inv[(4 * threadIdx.y + TILE_DIM * block_row) * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_0;
		inv[(4 * threadIdx.y + 1 + TILE_DIM * block_row) * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_1;
		inv[(4 * threadIdx.y + 2 + TILE_DIM * block_row) * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_2;
		inv[(4 * threadIdx.y + 3 + TILE_DIM * block_row) * n + (TILE_DIM * block_col + threadIdx.x)] = data.x_11_3;
	}
}

namespace mat
{
	/*
	* Compute inversion of square matrix 
	* 
	* This is GPU implementation of matrix inversion algorithm posted on www.mathworks.com
	* and can be found by link https://www.mathworks.com/matlabcentral/answers/243916-trying-to-write-a-program-that-calculates-the-inverse-of-a-3x3-matrix-my-program-works-for-some-mat#answer_324850
	* 
	* The original algorithm was modified so that the processing was performed in one pass.
	*/
	template<typename dataT>
	std::vector<dataT> Inversion(const float* in, int n)
	{
		dataT* d_data = nullptr;

		checkCudaErrors(hipMalloc((void**)&d_data, n * n * sizeof(dataT)));

		checkCudaErrors(hipMemcpy(d_data, in, n * n * sizeof(float), hipMemcpyHostToDevice));

		dataT* d_augmat = nullptr;

		checkCudaErrors(hipMalloc((void**)&d_augmat, n * 2 * n * sizeof(dataT)));

		dataT* d_buffer = nullptr;

		checkCudaErrors(hipMalloc((void**)&d_buffer, 4 * TILE_DIM * n * sizeof(dataT)));

		dim3 block_size(TILE_DIM, NUM_OF_WARPS_IN_BLOCK);

		hipFuncSetCacheConfig(reinterpret_cast<const void*>(mat_gpu::init_and_compute_echelon<dataT>), hipFuncCachePreferL1);

		hipFuncSetCacheConfig(reinterpret_cast<const void*>(mat_gpu::compute_echelon<dataT>), hipFuncCachePreferL1);

		const auto num_of_blocks_in_column = n / TILE_DIM;

		const auto num_of_processing_blocks = num_of_blocks_in_column * num_of_blocks_in_column + 1;

		for (auto iter = 0; iter < num_of_blocks_in_column; ++iter)
		{
			if (iter != 0)
			{
				mat_gpu::compute_echelon<dataT> <<< num_of_processing_blocks, block_size >>>( d_data
													    , d_augmat
													    , d_buffer
													    , n
													    , num_of_blocks_in_column
													    , iter);
			}
			else
			{
				mat_gpu::init_and_compute_echelon<dataT> <<< num_of_processing_blocks, block_size >>>( d_data
														     , d_augmat
														     , d_buffer
														     , n
														     , num_of_blocks_in_column
														     , iter);
			}

			const auto err = hipGetLastError();

			if (err != hipSuccess)
			{
				std::cerr << "Failed to launch kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;

				hipFree(d_data);

				hipFree(d_augmat);

				hipFree(d_buffer);

				return {};
			}

			hipDeviceSynchronize();
		}

		std::vector<dataT> I(n * n, 0.f);

		checkCudaErrors(hipMemcpy(I.data(), d_data, n * n * sizeof(dataT), hipMemcpyDeviceToHost));

		hipFree(d_data);

		hipFree(d_augmat);

		hipFree(d_buffer);

		return I;
	}

	template std::vector<float> Inversion<float>(const float* h_x, int n);
}

namespace mat_test
{
	void Inversion()
	{
		auto x = read_file<float>("test_data/2784_2784_matrix.csv", true);

		auto inv = mat::Inversion<float>(x.first.data(), 2784);

		assert(true == cmp<float>(inv, read_file<float>("test_data/2784_2784_inv_matrix.csv", false).first, 0.00000001f));

		x = read_file<float>("test_data/2048_2048_matrix.csv", true);
		
		inv = mat::Inversion<float>(x.first.data(), 2048);

		assert(true == cmp<float>(inv, read_file<float>("test_data/2048_2048_inv_matrix.csv", false).first, 0.000001f));

		x = read_file<float>("test_data/256_256_matrix.csv", true);
		
		inv = mat::Inversion<float>(x.first.data(), 256);
		
		assert(true == cmp<float>(inv, read_file<float>("test_data/256_256_inv_matrix.csv", false).first, 0.00000001f));

		x = read_file<float>("test_data/64_64_matrix.csv", true);
		
		inv = mat::Inversion<float>(x.first.data(), 64);

		assert(true == cmp<float>(inv, read_file<float>("test_data/64_64_inv_matrix.csv", false).first, 0.00000001f));
	}
}
